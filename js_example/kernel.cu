#include "hip/hip_runtime.h"
#include <GL/glut.h>
#include "math.h"
#include <stdlib.h>
#include <stdio.h>
#include <windows.h>
#include "vecter3.h"
#include "plane.h"
#include "sphere.h"
#include "perspectiveCamera.h"
#include <iostream>

using namespace std;

const GLdouble Pi = 3.1415926536;

static int day = 200; // day�ı仯����0��359

float pixels[4][10000000] = { 0 };

const float height = 1000;
const float width = 1000;


void init()
{
	glClearColor(1.0, 1.0, 1.0, 0.0);
}



//void myDisplay()
//{
//	int i;
//	// glShadeModel(GL_FLAT);
//	glClear(GL_COLOR_BUFFER_BIT);
//	glBegin(GL_TRIANGLE_FAN);
//	glColor3f(1.0f, 1.0f, 1.0f);
//	glVertex2f(0.0f, 0.0f);
//	for (i = 0; i <= 8; ++i)
//	{
//		glColor3f(i & 0x04, i & 0x02, i & 0x01);
//		glVertex2f(cos(i*Pi / 4), sin(i*Pi / 4));
//	}
//	glEnd();
//	glFlush();
//}


//void myDisplay()
//{
//	glClear(GL_COLOR_BUFFER_BIT);
//
//	glColor3ub(250,250,250);
//	glPointSize(10.0f);
//	glBegin(GL_POINTS);
//
//	/*glColor3ub(255,0,0);
//	glVertex2f(0.0f,0.0f);
//	glColor3ub(0,255,0);
//	glVertex2f(0.5f,0.5f);*/
//
//	int x = 0;
//	int y = 0;
//
//	for (x = 0; x <= 255; x++)
//	{
//		for (y = 0; y <= 255; y++)
//		{
//			glColor3ub(x,y,0);
//			glVertex2f(x/255.0f,y/255.0f);
//		}
//	
//	}
//
//	glEnd();
//	glFlush();
//}


void myDisplay()
{
	/*glClear(GL_COLOR_BUFFER_BIT);
	glColor3ub(255, 0, 255);
	glRectf(-0.5f, -0.5f, 0.5f, 0.5f);*/

	glClear(GL_COLOR_BUFFER_BIT);


	glColor3ub(0, 0, 0);
	//glPointSize(1.0f);
	glBegin(GL_POINTS);


	int x = 0;
	int y = 0;
	int i = 0;
	for (x = (-1)*height / 2; x <= height / 2; x++)
	{
		for (y = (-1)*width / 2; y <= width / 2; y++)
		{
			glColor3ub(pixels[0][i], pixels[1][i], pixels[2][i]);
			//glColor3ub(255, 1, 1);
			glVertex2f(y / width * 2, x / height * 2);
			i++;
		}

	}

	glEnd();
	glFlush();
}


IntersectResult intersect_fun(Plane plane1, Sphere sphere1, Sphere sphere2, Ray3 ray)
{
	float minDistance = 1000000.0;
	IntersectResult result = plane1.intersect(ray);
	IntersectResult minResult = result;
	if (result.geometry != 0 && result.distance < minDistance)
	{
		result.geometry = 1;
		minDistance = result.distance;
		minResult = result;
	}
	result = sphere1.intersect(ray);
	if (result.geometry != 0 && result.distance < minDistance)
	{
		result.geometry = 2;
		minDistance = result.distance;
		minResult = result;
	}
	result = sphere2.intersect(ray);
	if (result.geometry != 0 && result.distance < minDistance)
	{
		result.geometry = 3;
		minDistance = result.distance;
		minResult = result;
	}
	return minResult;
}

Color rayTraceRecursive(Plane plane1, Sphere sphere1, Sphere sphere2, Ray3 ray, float maxReflect)
{
	IntersectResult result = intersect_fun(plane1, sphere1, sphere2, ray);
	if (result.geometry != 0)
	{
		float reflectiveness;
		Color color;
		if (result.geometry == 1)
		{
			reflectiveness = plane1.material.reflectiveness;
			color = plane1.material.sample(ray, result.position, result.normal);
			color = color.multiply(1 - reflectiveness);
			if (reflectiveness > 0 && maxReflect > 0)
			{
				Vector3 r = result.normal.multiply((-2) * result.normal.dot(ray.direction)).add(ray.direction);
				Ray3 ray = Ray3(result.position, r);
				Color reflectiveColor = rayTraceRecursive(plane1, sphere1, sphere2, ray, maxReflect - 1);
				color = color.add(reflectiveColor.multiply(reflectiveness));
			}
		}
		if (result.geometry == 2)
		{
			reflectiveness = sphere1.material.reflectiveness;
			color = sphere1.material.sample(ray, result.position, result.normal);
			color = color.multiply(1 - reflectiveness);
			if (reflectiveness > 0 && maxReflect > 0)
			{
				Vector3 r = result.normal.multiply(-2 * result.normal.dot(ray.direction)).add(ray.direction);
				Ray3 ray = Ray3(result.position, r);
				Color reflectiveColor = rayTraceRecursive(plane1, sphere1, sphere2, ray, maxReflect - 1);
				color = color.add(reflectiveColor.multiply(reflectiveness));
			}
		}
		if (result.geometry == 3)
		{
			reflectiveness = sphere2.material.reflectiveness;
			color = sphere2.material.sample(ray, result.position, result.normal);
			color = color.multiply(1 - reflectiveness);
			if (reflectiveness > 0 && maxReflect > 0)
			{
				Vector3 r = result.normal.multiply(-2 * result.normal.dot(ray.direction)).add(ray.direction);
				Ray3 ray = Ray3(result.position, r);
				Color reflectiveColor = rayTraceRecursive(plane1, sphere1, sphere2, ray, maxReflect - 1);
				color = color.add(reflectiveColor.multiply(reflectiveness));
			}
		}
		return color;
	}
	else
		return Color::black();
}



void rayTraceReflection(Plane plane1, Sphere sphere1, Sphere sphere2, PerspectiveCamera perspective1, float maxReflect)
{
	//��ʼ��
	plane1.initialize();
	sphere1.initialize();
	sphere2.initialize();
	perspective1.initialize();

	int i = 0;
	for (int y = 0; y <= width; y++)
	{
		float sy = y / width;
		for (int x = 0; x <= height; x++)
		{
			float sx = 1 - x / height;
			Ray3 ray = perspective1.generateRay(sx, sy);
			Color color1 = rayTraceRecursive(plane1, sphere1, sphere2, ray, maxReflect);
			pixels[0][i] = color1.r * 255;
			pixels[1][i] = color1.g * 255;
			pixels[2][i] = color1.b * 255;
			i++;
		}
	}
}


int main(int argc, char *argv[])
{


	Plane plane1 = Plane(Vector3(0, 1, 0), 1);
	Sphere sphere1 = Sphere(Vector3(10, 10, -10), 10);
	Sphere sphere2 = Sphere(Vector3(-10, 10, -10), 10);
	plane1.material = CheckerMaterial(1, 0.5);
	sphere1.material = PhongMaterial(Color::red(), Color::white(), 16, 0.45);
	sphere2.material = PhongMaterial(Color::green(), Color::white(), 16, 0.55);
	PerspectiveCamera perspective1 = PerspectiveCamera(Vector3(0, 5, 20), Vector3(0, 0, -1), Vector3(0, 1, 0), 120);

	rayTraceReflection(plane1, sphere1, sphere2, perspective1, 1);

	//system("pause");
	glutInit(&argc, argv);								//��GLUT���г�ʼ��
	glutInitDisplayMode(GLUT_RGB | GLUT_SINGLE);
	//glutInitDisplayMode(GLUT_DOUBLE | GLUT_DEPTH | GLUT_RGBA | GLUT_MULTISAMPLE);
	//glEnable(GLUT_MULTISAMPLE);
	glutInitWindowPosition(100, 100);
	glutInitWindowSize(400, 400);
	glutCreateWindow("̫��");
	init();
	glutDisplayFunc(&myDisplay);
	glutMainLoop();

	/*int x = 0;
	int y = 0;
	int i = 0;
	for (x = 0; x <= 256; x++)
	{
		for (y = 0; y <= 256; y++)
		{
			printf("%f %f %f\n", pixels[0][i], pixels[1][i], pixels[2][i]);
		}
	}

	system("pause\n");*/
	return 0;
}